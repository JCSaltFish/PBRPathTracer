#include "hip/hip_runtime.h"
#include <glm/gtx/constants.hpp>

#include <hiprand/hiprand_kernel.h>

#include "cudakernel.cuh"

__constant__ hiprandState_t* state = 0;

__constant__ float PI;
__constant__ float EPSILON = 0.001f;

__constant__ int resX, resY;
__constant__ int maxDepth;
__constant__ int samples;

__constant__ float _camPos[3], _camDir[3], _camUp[3];
__constant__ float camFocal, camFovy;

__device__ GPUBVHNode* bvh = 0;
__constant__ int bvhSize;

struct GPUImage
{
	int width;
	int height;
	float* data;
	__host__ __device__ GPUImage() :
		width(0),
		height(0),
		data(0)
	{
	}
};
__device__ GPUImage* textures = 0;
int numTextures = 0;

__global__ void InitCuRand(int seed)
{
	const int x = threadIdx.x + blockIdx.x * blockDim.x;
	const int y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x >= resX || y >= resY)
		return;

	hiprand_init(seed, x + y * resX, 0, &state[x + y * resX]);
}

void InitCUDA()
{
	float h_pi = glm::pi<float>();
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(PI), &h_pi, sizeof(float)));
	gpuErrchk(hipDeviceSetLimit(hipLimitStackSize, 1024 * 8));
}

void CUDASetResolution(int x, int y)
{
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(resX), &x, sizeof(unsigned)));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(resY), &y, sizeof(unsigned)));

	if (state)
		gpuErrchk(hipFree(state));
	hiprandState_t* d_randState;
	gpuErrchk(hipMalloc((void**)&d_randState, x * y * sizeof(hiprandState_t)));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(state), &d_randState, sizeof(d_randState)));

	srand(time(0));
	int seed = rand();
	dim3 blockDim(16, 16, 1), gridDim(x / blockDim.x + 1, y / blockDim.y + 1, 1);
	InitCuRand << < gridDim, blockDim >> > (seed);
	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());
}

void CUDASetTraceDepth(int depth)
{
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(maxDepth), &depth, sizeof(unsigned)));
}

void CUDASetCamera(float* pos, float* dir, float* up)
{
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(_camPos), pos, sizeof(float) * 3));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(_camDir), dir, sizeof(float) * 3));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(_camUp), up, sizeof(float) * 3));
}

void CUDASetProjection(float f, float fovy)
{
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(camFocal), &f, sizeof(float)));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(camFovy), &fovy, sizeof(float)));
}

void CUDASetBVH(GPUBVHNode* nodes, int size)
{
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(bvhSize), &size, sizeof(unsigned)));

	BVHNode* d_Nodes;
	gpuErrchk(hipMalloc((void**)&d_Nodes, size * sizeof(GPUBVHNode)));
	gpuErrchk(hipMemcpy(d_Nodes, nodes, size * sizeof(GPUBVHNode), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(bvh), &d_Nodes, sizeof(GPUBVHNode*)));
}

void CUDALoadTextures(const std::vector<Image*>& texVec)
{
	int size = texVec.size();
	numTextures = size;

	GPUImage* h_normalImgs = new GPUImage[size];
	for (int i = 0; i < size; i++)
	{
		int w = texVec[i]->width();
		int h = texVec[i]->height();
		h_normalImgs[i].width = w;
		h_normalImgs[i].height = h;

		int size = w * h * 4 * sizeof(float);
		float* h_data = new float[size];
		memcpy(h_data, texVec[i]->data(), size);
		float* d_data;
		gpuErrchk(hipMalloc(&d_data, size));
		gpuErrchk(hipMemcpy(d_data, h_data, size, hipMemcpyHostToDevice));
		delete[] h_data;
		h_normalImgs->data = d_data;
	}

	GPUImage* d_normalImgs;
	gpuErrchk(hipMalloc(&d_normalImgs, size * sizeof(GPUImage)));
	gpuErrchk(hipMemcpy(d_normalImgs, h_normalImgs, size * sizeof(GPUImage), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(textures), &d_normalImgs, size * sizeof(GPUImage*)));
	delete[] h_normalImgs;
}

__device__ glm::vec4 CUDATex2D(const GPUImage& image, const glm::vec2& uv)
{
	if (uv.x > 1.0f || uv.x < 0.0f || uv.y > 1.0f || uv.y < 0.0f)
		return glm::vec4(0.0f);

	int w = image.width;
	int h = image.height;

	glm::ivec2 coord = glm::ivec2(w * uv.x, h * uv.y);
	float* p = image.data + (4 * (coord.y * w + coord.x));

	return glm::vec4(p[0], p[1], p[2], p[3]);
}

__device__ bool IsSameSide(const glm::vec3& p1, const glm::vec3& p2, const glm::vec3& a, const glm::vec3& b)
{
	glm::vec3 ba = b - a;
	glm::vec3 cp1 = glm::cross(ba, (p1 - a));
	glm::vec3 cp2 = glm::cross(ba, (p2 - a));

	return (glm::dot(cp1, cp2) >= 0);
}

__device__ bool IsInside(const glm::vec3& p, const glm::vec3& a, const glm::vec3& b, const glm::vec3& c)
{
	return (IsSameSide(p, a, b, c) && IsSameSide(p, b, a, c) && IsSameSide(p, c, a, b));
}

__device__ bool IntersectBox(const glm::vec3& ro, const glm::vec3& rd, const glm::vec3& bMin, const glm::vec3& bMax)
{
	glm::vec3 tMin = (bMin - ro) / rd;
	glm::vec3 tMax = (bMax - ro) / rd;
	glm::vec3 t1 = glm::min(tMin, tMax);
	glm::vec3 t2 = glm::max(tMin, tMax);
	float tNear = glm::max(glm::max(t1.x, t1.y), t1.z);
	float tFar = glm::min(glm::min(t2.x, t2.y), t2.z);
	if (tNear >= tFar)
		return false;
	return true;
}

__device__ bool Hit(const glm::vec3& ro, const glm::vec3& rd, Triangle& triangleOut, float& distOut)
{
	bool res = false;

	distOut = float(0xFFFF);

	GPUBVHNode* stack[64];
	GPUBVHNode** pStack = stack;
	*pStack++ = NULL;

	GPUBVHNode* currNode = bvh;
	int stackIndex = 1;
	do
	{
		if (IntersectBox(ro, rd, currNode->box.min, currNode->box.max))
		{
			if (currNode->rightOffset == -1) // leaf
			{
				if (glm::dot(rd, currNode->triangle.normal) != 0.0f)
				{
					float d = glm::dot((currNode->triangle.v1 - ro), currNode->triangle.normal) / glm::dot(rd, currNode->triangle.normal);
					if (d >= 0)
					{
						glm::vec3 p = ro + rd * d;
						if (IsInside(p, currNode->triangle.v1, currNode->triangle.v2, currNode->triangle.v3))
						{
							if (d < distOut)
							{
								distOut = d;
								triangleOut = currNode->triangle;
							}
							res = true;
						}
					}
				}
				currNode = *--pStack;
				stackIndex--;
			}
			else // interier
			{
				GPUBVHNode* left = &(bvh[currNode->nodeIndex + 1]);
				GPUBVHNode* right = &(bvh[currNode->nodeIndex + currNode->rightOffset]);
				currNode = left;
				*pStack++ = right;
				stackIndex++;
			}
		}
		else
		{
			currNode = *--pStack;
			stackIndex--;
		}
	} while (stackIndex > 0 && stackIndex < 64);

	return res;
}

__device__ glm::vec2 GetUV(const glm::vec3& p, const Triangle& t)
{
	glm::vec3 v2 = p - t.v1;
	float d20 = glm::dot(v2, t.barycentricInfo.v0);
	float d21 = glm::dot(v2, t.barycentricInfo.v1);

	float alpha = (t.barycentricInfo.d11 * d20 - t.barycentricInfo.d01 * d21) *
		t.barycentricInfo.invDenom;
	float beta = (t.barycentricInfo.d00 * d21 - t.barycentricInfo.d01 * d20) *
		t.barycentricInfo.invDenom;

	return (1.0f - alpha - beta) * t.uv1 + alpha * t.uv2 + beta * t.uv3;
}

__device__ glm::vec3 GetSmoothNormal(const glm::vec3& p, const Triangle& t)
{
	glm::vec3 v2 = p - t.v1;
	float d20 = glm::dot(v2, t.barycentricInfo.v0);
	float d21 = glm::dot(v2, t.barycentricInfo.v1);

	float alpha = (t.barycentricInfo.d11 * d20 - t.barycentricInfo.d01 * d21) *
		t.barycentricInfo.invDenom;
	float beta = (t.barycentricInfo.d00 * d21 - t.barycentricInfo.d01 * d20) *
		t.barycentricInfo.invDenom;

	glm::vec3 n = (1.0f - alpha - beta) * t.n1 + alpha * t.n2 + beta * t.n3;
	glm::vec3 res = glm::normalize(glm::vec3(n.x, -n.y, n.z));
	return glm::normalize(n);
}

__device__ glm::vec3 reflect(glm::vec3 I, glm::vec3 N)
{
	return I - N * glm::dot(N, I) * glm::vec3(2);
}

__device__ glm::vec3 Trace(glm::vec3 ro, glm::vec3 rd, int& depth, bool& inside, hiprandState_t& state)
{
	float d = 0.0f;
	Triangle t;
	if (Hit(ro, rd, t, d))
	{
		Material& mat = t.material;
		glm::vec3 p = ro + rd * d;
		glm::vec2 uv = GetUV(p, t);
		glm::vec3 n = t.normal;
		if (t.smoothing)
			n = GetSmoothNormal(p, t);
		if (glm::dot(n, rd) > 0.0f)
			n = -n;
		if (mat.normalTexId != -1)
		{
			glm::mat3 TBN = glm::mat3(t.tangent, t.bitangent, n);
			glm::vec3 nt = glm::vec3(CUDATex2D(textures[mat.normalTexId], uv)) * 2.0f - 1.0f;
			if (nt.z < 0.0f)
				nt = glm::vec3(nt.x, nt.y, 0.0f);
			nt = glm::normalize(nt);
			n = glm::normalize(TBN * nt);
		}
		p += n * EPSILON;

		if (depth < maxDepth * 2)
		{
			depth++;
			// Russian Roulette Path Termination
			float prob = glm::min(0.95f, glm::max(glm::max(mat.baseColor.x, mat.baseColor.y), mat.baseColor.z));
			if (depth >= maxDepth)
			{
				if (fabs(hiprand_uniform(&state)) > prob)
					return mat.emissive * mat.emissiveIntensity;
			}

			glm::vec3 r = reflect(rd, n);
			glm::vec3 reflectDir = r;
			if (mat.type == MaterialType::SPECULAR)
				reflectDir = r;
			else if (mat.type == MaterialType::DIFFUSE)
			{
				// Monte Carlo Integration
				glm::vec3 u = glm::abs(n.x) < 1.0f - EPSILON ? glm::cross(glm::vec3(1.0f, 0.0f, 0.0f), n) : glm::cross(glm::vec3(1.0f), n);
				u = glm::normalize(u);
				glm::vec3 v = glm::normalize(glm::cross(u, n));
				float w = hiprand_uniform(&state), theta = hiprand_uniform(&state);
				// uniformly sampling on hemisphere
				reflectDir = w * cosf(2.0f * PI * theta) * u + w * sinf(2.0f * PI * theta) * v + glm::sqrt(1.0f - w * w) * n;
				reflectDir = glm::normalize(reflectDir);
			}
			else if (mat.type == MaterialType::GLOSSY)
			{
				// Monte Carlo Integration
				glm::vec3 u = fabs(n.x) < 1 - FLT_EPSILON ? glm::cross(glm::vec3(1, 0, 0), r) : glm::cross(glm::vec3(1), r);
				u = glm::normalize(u);
				glm::vec3 v = glm::cross(u, r);
				float w = hiprand_uniform(&state) * mat.roughness, theta = hiprand_uniform(&state);
				// wighted sampling on hemisphere
				reflectDir = w * cosf(2 * PI * theta) * u + w * sinf(2 * PI * theta) * v + sqrtf(1 - w * w) * r;
			}
			else if (mat.type == MaterialType::GLASS)
			{
				float nc = 1.0f, ng = 1.5f;
				// Snells law
				float eta = inside ? ng / nc : nc / ng;
				float r0 = (nc - ng) / (nc + ng);
				r0 = r0 * r0;
				float c = fabs(glm::dot(rd, n));
				float k = 1.0f - eta * eta * (1.0f - c * c);
				if (k < 0.0f)
					reflectDir = r;
				else
				{
					// Shilick's approximation of Fresnel's equation
					float re = r0 + (1.0f - r0) * (1.0f - c) * (1.0f - c);
					if (fabs(hiprand_uniform(&state)) < re)
						reflectDir = r;
					else
					{
						reflectDir = glm::normalize(eta * rd - (eta * glm::dot(n, rd) + sqrtf(k)) * n);
						p -= n * EPSILON * 2.0f;
						inside = !inside;
					}
				}
			}
			
			return mat.emissive * mat.emissiveIntensity + Trace(p, reflectDir, depth, inside, state) * mat.baseColor;
		}
	}

	return glm::vec3(0.0f);
}

__global__ void RenderPixel(float* img)
{
	const int x = threadIdx.x + blockIdx.x * blockDim.x;
	const int y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x >= resX || y >= resY)
		return;
	const int index = x + (resY - y - 1)*resX;
	hiprandState_t localState = state[index];

	// Position world space image plane
	glm::vec3 camPos = glm::vec3(_camPos[0], _camPos[1], _camPos[2]);
	glm::vec3 camDir = glm::vec3(_camDir[0], _camDir[1], _camDir[2]);
	glm::vec3 camUp = glm::vec3(_camUp[0], _camUp[1], _camUp[2]);
	glm::vec3 imgCenter = camPos + camDir * camFocal;
	float imgHeight = 2.0f * camFocal * tan((camFovy / 2.0f) * PI / 180.0f);
	float aspect = (float)resX / (float)resY;
	float imgWidth = imgHeight * aspect;
	float deltaX = imgWidth / (float)resX;
	float deltaY = imgHeight / (float)resY;
	glm::vec3 camRight = glm::normalize(glm::cross(camUp, camDir));

	// Starting at top left
	glm::vec3 topLeft = imgCenter - camRight * (imgWidth * 0.5f);
	topLeft += camUp * (imgHeight * 0.5f);
	glm::vec3 pixel = topLeft - camUp * (float(y) * deltaY) + camRight * (float(x) * deltaX);

	glm::vec3 rayDir = glm::normalize(pixel - camPos);
	int depth = 0;
	bool inside = false;
	glm::vec3 color = Trace(camPos, rayDir, depth, inside, localState);

	// Draw
	glm::vec3 preColor = glm::vec3();
	memcpy(&preColor[0], img + 3 * index, 3 * sizeof(float));
	color = (preColor * float(samples - 1) + color) / float(samples);
	color = glm::clamp(color, glm::vec3(0.0f), glm::vec3(1.0f));
	memcpy(img + 3 * index, &color[0], 3 * sizeof(float));
	state[index] = localState;
}

void CUDARenderFrame(int w, int h, float* img, int& h_samples)
{
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(samples), &h_samples, sizeof(int)));
	dim3 blockDim(16, 16, 1), gridDim(w / blockDim.x + 1, h / blockDim.y + 1, 1);
	RenderPixel << <gridDim, blockDim >> > (img);
	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());
}

void CUDAReset()
{
	if (bvh != 0)
	{
		gpuErrchk(hipFree(bvh));
		bvh = 0;
	}

	if (textures != 0)
	{
		for (int i = 0; i < numTextures; i++)
			gpuErrchk(hipFree(textures[i].data));
		gpuErrchk(hipFree(textures));
		textures = 0;
		numTextures = 0;
	}
}

void CUDAFinish()
{
	CUDAReset();
	if (state)
		gpuErrchk(hipFree(state));
}
